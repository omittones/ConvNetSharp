
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, double* __restrict input, double* __restrict output) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) output[i] =  input[i] > 0 ? input[i] : input[i] * 0.01; 
	}
}